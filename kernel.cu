
#include "hip/hip_runtime.h"

#include <hip/device_functions.h>
#include <stdio.h>
#include <cstdlib>
#include <math.h>
#include<time.h>



#include <hip/hip_runtime_api.h>




#define Number 1000
#define Delta_t 0.01

__global__
void Simulate(double* Vortex_p, double* Omega_v_p, double* VortexN_p, double* Omega_vN_p, double *Sigma_p)
{
	double radiika_p, t1_p, t2_p;
		double t3_p, Om22P_p, ssss_p, vxx_p, vyy_p, vzz_p;
		double dvxdxmov, dvxdymov, dvxdzmov;
		double  dvydxmov, dvydymov, dvydzmov;
		double dvzdxmov,  dvzdymov,  dvzdzmov;

	double Vxc, Vyc, Vzc, dssss_dr;
	
	int j =  3*(threadIdx.x + blockIdx.x*blockDim.x);

	if (true) {
		Vxc = 1.0;
		Vyc = 0;
		Vzc = 0;
		dvxdxmov = 0.0;
		dvxdymov = 0;
		dvxdzmov = 0;

		dvydxmov = 0;
		dvydymov = 0;
		dvydzmov = 0;

		dvzdxmov = 0;
		dvzdymov = 0;
		dvzdzmov = 0;

	}
	
	for (int i=0; i < Number; i++) {
		vxx_p = Vortex_p[j] - Vortex_p[i * 3];
		vyy_p = Vortex_p[j + 1] - Vortex_p[(i * 3) + 1];
		vzz_p = Vortex_p[j + 2] - Vortex_p[(i * 3) + 2];
		radiika_p = vxx_p*vxx_p + vyy_p*vyy_p + vzz_p*vzz_p;
		t1_p = vyy_p*Omega_v_p[(i * 3) + 2] - vzz_p*Omega_v_p[(i * 3) + 1];
		t2_p = vzz_p*Omega_v_p[i * 3] - vxx_p*Omega_v_p[(i * 3) + 2];
		t3_p = vxx_p*Omega_v_p[(i * 3) + 1] - vyy_p*Omega_v_p[i * 3];
		Om22P_p = 3.1416 / Sigma_p[i] / Sigma_p[i] / 2.0;
		ssss_p = exp(-radiika_p*Om22P_p);

		Vxc = Vxc + ssss_p*t1_p;
		Vyc = Vyc + ssss_p*t2_p;
		Vzc = Vzc + ssss_p*t3_p;

		dssss_dr = (-Om22P_p)*ssss_p;

		dvxdxmov = dssss_dr*vxx_p*t1_p + dvxdxmov;
		dvxdymov = dssss_dr*vyy_p*t1_p + Omega_v_p[(i * 3) + 2] * ssss_p + dvxdymov;
		dvxdzmov = dssss_dr*vzz_p*t1_p - Omega_v_p[(i * 3) + 1] * ssss_p + dvxdzmov;

		dvydxmov = dssss_dr*vxx_p*t2_p - Omega_v_p[(i * 3) + 2] * ssss_p + dvydxmov;
		dvydymov = dssss_dr*vyy_p*t2_p + dvydymov;
		dvydzmov = dssss_dr*vzz_p*t2_p + Omega_v_p[i * 3] * ssss_p + dvydzmov;

		dvzdxmov = dssss_dr*vxx_p*t3_p + Omega_v_p[(i * 3) + 1] * ssss_p + dvzdxmov;
		dvzdymov = dssss_dr*vyy_p*t3_p - Omega_v_p[i * 3] * ssss_p + dvzdymov;
		dvzdzmov = dssss_dr*vzz_p*t3_p + dvzdzmov;
	}

	if ( true) {
		
		VortexN_p[j] = Vortex_p[j] + Delta_t*Vxc;
		VortexN_p[j + 1] = Vortex_p[j + 1] + Delta_t*Vyc;
		VortexN_p[j + 2] = Vortex_p[j + 2] + Delta_t*Vzc;

		

		double domxdt, domydt, domzdt;
		domxdt = dvxdxmov*Omega_v_p[j] + dvxdymov*Omega_v_p[j + 1] + dvxdzmov*Omega_v_p[j + 2];
		domydt = dvydxmov*Omega_v_p[j] + dvydymov*Omega_v_p[j + 1] + dvydzmov*Omega_v_p[j + 2];
		domzdt = dvzdxmov*Omega_v_p[j] + dvzdymov*Omega_v_p[j + 1] + dvzdzmov*Omega_v_p[j + 2];


		Omega_vN_p[j] = Omega_v_p[j] + domxdt*Delta_t;
		Omega_vN_p[j + 1] = Omega_v_p[j + 1] + domydt*Delta_t;
		Omega_vN_p[j + 2] = Omega_v_p[j + 2] + domzdt*Delta_t;
		Vxc = 0, Vyc = 0, Vzc = 0;
		dvxdxmov = 0, dvxdymov = 0, dvxdzmov = 0;
		dvydxmov = 0, dvydymov = 0, dvydzmov = 0;
		dvzdxmov = 0, dvzdymov = 0, dvzdzmov = 0;

	}
		
	
	
}





int main()
{

	const int Ntime = 10000;
	//const double Delta_t = 0.01;
	const double Radius = 0.1;
	//const int Number = 10;
	//const double V_mean = 1.0;
	double Vortex[Number][3];
	double Omega_v[Number][3];
	double VortexN[Number][3];
	double Omega_vN[Number][3];
	
		double *Vortex_p=new double[Number*3];
		double *Omega_v_p = new double[Number * 3];
		double *VortexN_p = new double[Number * 3];
		double *Omega_vN_p = new double[Number * 3];

	double Sigma[Number];
	double *Sigma_p;
	double StatisticalMoments[4] = {0.000};
	double Amagni=0.0,Amagnit_old,Amagnit_new,Speed_max,Sigmas;
	double Energy=0;
	int Ncout=0;
	FILE *fp1,*fp2;
	fp1 = fopen("D:\\cudaa\\Velocities1.txt","w+");
	fp2 = fopen("D:\\cudaa\\MaxValue1.txt","w+");
	double Vx;
	double vxx, vyy, vzz;
	double *vxx_p, *vyy_p, *vzz_p;
	clock_t time0,time1;
	for (int ivorton = 0; ivorton < Number; ivorton++) {
		
		Vortex[ivorton][0] = (double)rand() / (double)RAND_MAX;
		Vortex[ivorton][1] = (double)rand() / (double)RAND_MAX;
		Vortex[ivorton][2] = (double)rand() / (double)RAND_MAX;

		Omega_v[ivorton][0] = (((double)rand() / (double)RAND_MAX) - 0.5);
		Omega_v[ivorton][1] = (((double)rand() / (double)RAND_MAX) - 0.5);
		Omega_v[ivorton][2] = (((double)rand() / (double)RAND_MAX) - 0.5);
		Sigma[ivorton] = Radius;
		

		//printf("%f",Vortex[ivorton][1]);

	}

	int counter = 0;
	for (int h = 0; h < Number; h++) {
		for (int w = 0; w < 3; w++) {
			Vortex_p[counter] = Vortex[h][w];
			Omega_v_p[counter] = Omega_v[h][w];
			VortexN_p[counter] = VortexN[h][w];
			Omega_vN_p[counter] = Omega_vN[h][w];
			counter++;
		}
	}


	




	time0 = clock();
	double radiika;
	double t1, t2, t3;
	double Om22P;
	double ssss,dssss_dr;
	double domxdt, domydt, domzdt,Replace=0;
	double Vxc, Vyc , Vzc ;
	double dvxdxmov , dvxdymov , dvxdzmov ;
	double  dvydxmov , dvydymov , dvydzmov ;
	double dvzdxmov , dvzdymov , dvzdzmov ;

	




	double *radiika_p;
	double *t1_p, *t2_p, *t3_p;
	double *Om22P_p;
	double *ssss_p, *dssss_dr_p;
	double *domxdt_p, *domydt_p, *domzdt_p, *Replace_p ;
	double *Vxc_p, *Vyc_p, *Vzc_p;
	double *dvxdxmov_p, *dvxdymov_p, *dvxdzmov_p;
	double  *dvydxmov_p, *dvydymov_p, *dvydzmov_p;
	double *dvzdxmov_p, *dvzdymov_p, *dvzdzmov_p;
	vxx_p = &vxx;
	hipMalloc((void**)&Vortex_p, (Number * 3) * sizeof(double));
	hipMalloc((void**)&Omega_v_p, (Number * 3) * sizeof(double));
	hipMalloc((void**)&VortexN_p, (Number * 3) * sizeof(double));
	hipMalloc((void**)&Omega_vN_p, (Number * 3) * sizeof(double));
	hipMalloc((void**)&Sigma_p, (Number) * sizeof(double));
	

	

	for (int itime = 0; itime < Ntime; itime++) {
		printf("%*d %f %e %d \n ",4,itime,Amagni,Energy,Ncout);
		//cudaMalloc((void**)&domzdt_p, sizeof(double));

		hipMemcpy(Vortex_p, Vortex, (Number * 3) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(Omega_v_p, Omega_v, (Number * 3) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(VortexN_p, VortexN, (Number * 3) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(Omega_vN_p, Omega_vN, (Number * 3) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(Sigma_p, Sigma, (Number) * sizeof(double), hipMemcpyHostToDevice);
		

		dim3 dimBlock(Number, 1);
		dim3 dimGrid(Number, 1);


		Simulate <<<Number/32, 32 >>> (Vortex_p,Omega_v_p, VortexN_p, Omega_vN_p,Sigma_p);

		hipMemcpy(Vortex, Vortex_p,  (Number * 3) * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(Omega_v,Omega_v_p, (Number * 3) * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(VortexN, VortexN_p,  (Number * 3) * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(Omega_vN, Omega_vN_p,  (Number * 3) * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(Sigma, Sigma_p,  (Number) * sizeof(double), hipMemcpyDeviceToHost);
		
	




		/*
		for (int ivorton = 0; ivorton < Number;ivorton++) {
			double Vxc=V_mean, Vyc=0.0, Vzc=0.0;
			double dvxdxmov = 0.0, dvxdymov = 0.0, dvxdzmov = 0.0;
			double  dvydxmov = 0.0, dvydymov = 0.0, dvydzmov = 0.0;
			double dvzdxmov = 0.0, dvzdymov = 0.0, dvzdzmov = 0.0;

			for (int induced = 0; induced < Number; induced++) {
				 vxx = Vortex[ivorton][0] - Vortex[induced][0];
				 vyy = Vortex[ivorton][1] - Vortex[induced][1];
				 vzz = Vortex[ivorton][2] - Vortex[induced][2];
				radiika = vxx*vxx + vyy*vyy + vzz*vzz;
				t1 = vyy*Omega_v[induced][2] - vzz*Omega_v[induced][1];
				t2 = vzz*Omega_v[induced][0] - vzz*Omega_v[induced][2];
				t3 = vxx*Omega_v[induced][1] - vzz*Omega_v[induced][0];
				Om22P = 3.1416 / Sigma[induced] / Sigma[induced] / 2.0;
				ssss = exp(-radiika* Om22P);
				 
				Vxc = Vxc + ssss*t1;
				Vyc = Vyc + ssss*t2;
				Vzc = Vzc + ssss*t3;

				dssss_dr = (-Om22P)*ssss;

				dvxdxmov = dssss_dr*vxx*t1 + dvxdxmov;
				dvxdymov = dssss_dr*vyy*t1 + Omega_v[induced][2]*ssss + dvxdymov;
				dvxdzmov = dssss_dr*vzz*t1 - Omega_v[induced][1]*ssss + dvxdzmov;

				dvydxmov = dssss_dr*vxx*t2 - Omega_v[induced][2]*ssss + dvydxmov;
				dvydymov = dssss_dr*vyy*t2 + dvydymov;
				dvydzmov = dssss_dr*vzz*t2 + Omega_v[induced][0]*ssss + dvydzmov;

				dvzdxmov = dssss_dr*vxx*t3 + Omega_v[induced][1]*ssss + dvzdxmov;
				dvzdymov = dssss_dr*vyy*t3 - Omega_v[induced][0]*ssss + dvzdymov;
				dvzdzmov = dssss_dr*vzz*t3 + dvzdzmov;




			}

			VortexN[ivorton][0] = Vortex[ivorton][0] + Delta_t*Vxc;
			VortexN[ivorton][1] = Vortex[ivorton][1] + Delta_t*Vyc;
			VortexN[ivorton][2] = Vortex[ivorton][2] + Delta_t*Vzc;

		//	domxdt=dvxdxmov*Omega_v[ivorton][0]+dvxdymov*Omega_v[][]

			domxdt = dvxdxmov*Omega_v[ivorton][0] + dvxdymov*Omega_v[ivorton][1] + dvxdzmov*Omega_v[ivorton][2];
			domydt = dvydxmov*Omega_v[ivorton][0] + dvydymov*Omega_v[ivorton][1] + dvydzmov*Omega_v[ivorton][2];
			domzdt = dvzdxmov*Omega_v[ivorton][0] + dvzdymov*Omega_v[ivorton][1] + dvzdzmov*Omega_v[ivorton][2];
			Omega_vN[ivorton][0] = Omega_v[ivorton][0] + domxdt*Delta_t;
			Omega_vN[ivorton][1]= Omega_v[ivorton][1] + domydt*Delta_t;
			Omega_vN[ivorton][2] = Omega_v[ivorton][2] + domzdt*Delta_t;



		}
		*/
		Ncout = 0;
		for (int ivorton = 0; ivorton < Number; ivorton++) {
			Replace = 0.0;
			for (int kkk = 0; kkk < 3; kkk++) {
				if (VortexN[ivorton][kkk] < 0.0) {
					Replace = 1.0;
				}
				if (VortexN[ivorton][kkk] > 1.0) {
					Replace = 1.0;
				}

			}
			if (Replace == 1.0) {
				Ncout = Ncout + 1;

				VortexN[ivorton][0] = (double)rand() / (double)RAND_MAX;
				VortexN[ivorton][1] = (double)rand() / (double)RAND_MAX;
				VortexN[ivorton][2] = (double)rand() / (double)RAND_MAX;

				Omega_vN[ivorton][0] = (((double)rand() / (double)RAND_MAX) - 0.5);
				Omega_vN[ivorton][1] = (((double)rand() / (double)RAND_MAX) - 0.5);
				Omega_vN[ivorton][2] = (((double)rand() / (double)RAND_MAX) - 0.5);
				Sigma[ivorton] = Radius;

			}
		}
		Amagni = 0.0;
		for (int ivorton = 0; ivorton < Number; ivorton++) {
			Vortex[ivorton][0] = VortexN[ivorton][0];
			Vortex[ivorton][1] = VortexN[ivorton][1];
			Vortex[ivorton][2] = VortexN[ivorton][2];

			Amagnit_old = sqrt((Omega_v[ivorton][0] * Omega_v[ivorton][0]) +( Omega_v[ivorton][1] * Omega_v[ivorton][1]) +( Omega_v[ivorton][2] * Omega_v[ivorton][2]));

			Omega_v[ivorton][0] = Omega_vN[ivorton][0];
			Omega_v[ivorton][1] = Omega_vN[ivorton][1];
			Omega_v[ivorton][2] = Omega_vN[ivorton][2];

			Amagnit_new= sqrt((Omega_v[ivorton][0] * Omega_v[ivorton][0]) + (Omega_v[ivorton][1] * Omega_v[ivorton][1]) + (Omega_v[ivorton][2] * Omega_v[ivorton][2]));
			Sigma[ivorton] = Sigma[ivorton] * sqrt(Amagnit_old / Amagnit_new);
			if (Amagnit_new >= Amagni) {
				Amagni = Amagnit_new;
				Energy = (Amagnit_new*Amagnit_new)*(pow(Sigma[ivorton], 5));
				Speed_max = Amagnit_new*Sigma[ivorton];
				Sigmas = Sigma[ivorton];
			}
		}

		//file write
		fprintf(fp1, "%f %f %f %f %f \n", itime*Delta_t, Amagni, Energy, Speed_max, Sigmas);
		Vx = 0.0;
		for (int induced = 0; induced < Number; induced++) {
			vxx = 0.5 - Vortex[induced][0];
			vyy = 0.5 - Vortex[induced][1];
			vzz = 0.5 - Vortex[induced][2];
			radiika = vxx*vxx + vyy*vyy + vzz*vzz;
			t1 = vyy*	Omega_v[induced][2] - vzz*Omega_v[induced][1];
			Om22P = 3.1416 / Sigma[induced] / Sigma[induced] / 2.0;
			ssss = exp(-radiika*Om22P);
			Vx = Vx + ssss*t1;

		}
		fprintf(fp2, "%f %f \n", itime*Delta_t, Vx);
		for (int ier = 0; ier < 4; ier++) {
			StatisticalMoments[ier] = StatisticalMoments[ier]  +pow(Vx, ier);
		}
		
	}
		hipFree(Vortex_p);
		hipFree(Omega_v_p);
		hipFree(VortexN_p);
		hipFree(Omega_vN_p);
		hipFree(Sigma_p);
		

	fclose(fp1);
	fclose(fp2);
	time1 = clock();
	printf("Time taken for execution with GPU acceleration (~31blocks & 32 threads)= %f sec(s)", (double)(time1 - time0) / CLOCKS_PER_SEC);

    
}

